#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cudaOperations.h"
#include <stdio.h>
#include <stdlib.h>

#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Grayscale conversion kernel
__global__ void grayscaleKernel(const unsigned char* input, unsigned char* output, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = (y * width + x) * channels;
        unsigned char r = input[idx];
        unsigned char g = input[idx + 1];
        unsigned char b = input[idx + 2];
        output[y * width + x] = static_cast<unsigned char>(0.299f * r + 0.587f * g + 0.114f * b);
    }
}

extern "C" void convertToGrayscaleCUDA(const unsigned char* h_input, unsigned char* h_output, int width, int height, int channels) {
    unsigned char *d_input, *d_output;
    size_t colorBytes = width * height * channels * sizeof(unsigned char);
    size_t grayBytes = width * height * sizeof(unsigned char);

    cudaCheckError(hipMalloc(&d_input, colorBytes));
    cudaCheckError(hipMalloc(&d_output, grayBytes));
    cudaCheckError(hipMemcpy(d_input, h_input, colorBytes, hipMemcpyHostToDevice));

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
    grayscaleKernel<<<grid, block>>>(d_input, d_output, width, height, channels);
    cudaCheckError(hipDeviceSynchronize());

    cudaCheckError(hipMemcpy(h_output, d_output, grayBytes, hipMemcpyDeviceToHost));
    hipFree(d_input);
    hipFree(d_output);
}

// Nearest-neighbor image resize kernel
__global__ void resizeKernel(const unsigned char* input, unsigned char* output,
                             int inWidth, int inHeight, int channels,
                             int outWidth, int outHeight) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < outWidth && y < outHeight) {
        float scaleX = static_cast<float>(inWidth) / outWidth;
        float scaleY = static_cast<float>(inHeight) / outHeight;
        int srcX = min(static_cast<int>(x * scaleX), inWidth - 1);
        int srcY = min(static_cast<int>(y * scaleY), inHeight - 1);
        int srcIdx = (srcY * inWidth + srcX) * channels;
        int dstIdx = (y * outWidth + x) * channels;
        for (int c = 0; c < channels; c++) {
            output[dstIdx + c] = input[srcIdx + c];
        }
    }
}

extern "C" void resizeImageCUDA(const unsigned char* h_input, unsigned char* h_output,
                                  int inWidth, int inHeight, int channels,
                                  int outWidth, int outHeight) {
    unsigned char *d_input, *d_output;
    size_t inBytes = inWidth * inHeight * channels * sizeof(unsigned char);
    size_t outBytes = outWidth * outHeight * channels * sizeof(unsigned char);

    cudaCheckError(hipMalloc(&d_input, inBytes));
    cudaCheckError(hipMalloc(&d_output, outBytes));
    cudaCheckError(hipMemcpy(d_input, h_input, inBytes, hipMemcpyHostToDevice));

    dim3 block(16, 16);
    dim3 grid((outWidth + block.x - 1) / block.x, (outHeight + block.y - 1) / block.y);
    resizeKernel<<<grid, block>>>(d_input, d_output, inWidth, inHeight, channels, outWidth, outHeight);
    cudaCheckError(hipDeviceSynchronize());

    cudaCheckError(hipMemcpy(h_output, d_output, outBytes, hipMemcpyDeviceToHost));
    hipFree(d_input);
    hipFree(d_output);
}

// Convolution kernel
__global__ void convolutionKernel(const unsigned char* input, unsigned char* output,
                                  int width, int height, int channels,
                                  const float* kernel, int kernelSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int kHalf = kernelSize / 2;
    if (x < width && y < height) {
        for (int c = 0; c < channels; c++) {
            float sum = 0.0f;
            for (int ky = -kHalf; ky <= kHalf; ky++) {
                for (int kx = -kHalf; kx <= kHalf; kx++) {
                    int ix = min(max(x + kx, 0), width - 1);
                    int iy = min(max(y + ky, 0), height - 1);
                    int imgIdx = (iy * width + ix) * channels + c;
                    int kIdx = (ky + kHalf) * kernelSize + (kx + kHalf);
                    sum += kernel[kIdx] * static_cast<float>(input[imgIdx]);
                }
            }
            int pixel = min(max(static_cast<int>(sum), 0), 255);
            output[(y * width + x) * channels + c] = static_cast<unsigned char>(pixel);
        }
    }
}

extern "C" void applyConvolutionCUDA(const unsigned char* h_input, unsigned char* h_output,
                                       int width, int height, int channels,
                                       const float* h_kernel, int kernelSize) {
    unsigned char *d_input, *d_output;
    float *d_kernel;
    size_t imgBytes = width * height * channels * sizeof(unsigned char);
    size_t kernelBytes = kernelSize * kernelSize * sizeof(float);

    cudaCheckError(hipMalloc(&d_input, imgBytes));
    cudaCheckError(hipMalloc(&d_output, imgBytes));
    cudaCheckError(hipMalloc(&d_kernel, kernelBytes));
    cudaCheckError(hipMemcpy(d_input, h_input, imgBytes, hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(d_kernel, h_kernel, kernelBytes, hipMemcpyHostToDevice));

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
    convolutionKernel<<<grid, block>>>(d_input, d_output, width, height, channels, d_kernel, kernelSize);
    cudaCheckError(hipDeviceSynchronize());

    cudaCheckError(hipMemcpy(h_output, d_output, imgBytes, hipMemcpyDeviceToHost));
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_kernel);
}

// Grayscale-to-Color Conversion using Pseudo‑Color Mapping Kernel
__global__ void grayscaleToPseudoColorKernel(const unsigned char* gray, unsigned char* color, const unsigned char* lut, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = y * width + x;
        unsigned char intensity = gray[idx];
        int lutIdx = intensity * 3;
        int colorIdx = idx * 3;
        color[colorIdx]     = lut[lutIdx];      
        color[colorIdx + 1] = lut[lutIdx + 1];     
        color[colorIdx + 2] = lut[lutIdx + 2];   
    }
}

extern "C" void grayscaleToColorCUDA(const unsigned char* h_gray, unsigned char* h_color, int width, int height) {
    unsigned char *d_gray, *d_color, *d_lut;
    size_t grayBytes = width * height * sizeof(unsigned char);
    size_t colorBytes = width * height * 3 * sizeof(unsigned char);
    size_t lutBytes = 256 * 3 * sizeof(unsigned char);
    unsigned char h_lut[256 * 3];
    for (int i = 0; i < 256; i++) {
        float normalized = i / 255.0f;
        float r, g, b;
        if (normalized < 0.33f) {
            r = 0;
            g = 255 * (normalized / 0.33f);
            b = 255;
        } else if (normalized < 0.66f) {
            r = 255 * ((normalized - 0.33f) / 0.33f);
            g = 255;
            b = 255 * (1 - ((normalized - 0.33f) / 0.33f));
        } else {
            r = 255;
            g = 255 * (1 - ((normalized - 0.66f) / 0.34f));
            b = 0;
        }
        h_lut[i * 3]     = static_cast<unsigned char>(b);
        h_lut[i * 3 + 1] = static_cast<unsigned char>(g);
        h_lut[i * 3 + 2] = static_cast<unsigned char>(r);
    }

    cudaCheckError(hipMalloc(&d_gray, grayBytes));
    cudaCheckError(hipMalloc(&d_color, colorBytes));
    cudaCheckError(hipMalloc(&d_lut, lutBytes));
    cudaCheckError(hipMemcpy(d_gray, h_gray, grayBytes, hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(d_lut, h_lut, lutBytes, hipMemcpyHostToDevice));

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
    grayscaleToPseudoColorKernel<<<grid, block>>>(d_gray, d_color, d_lut, width, height);
    cudaCheckError(hipDeviceSynchronize());

    cudaCheckError(hipMemcpy(h_color, d_color, colorBytes, hipMemcpyDeviceToHost));
    hipFree(d_gray);
    hipFree(d_color);
    hipFree(d_lut);
}